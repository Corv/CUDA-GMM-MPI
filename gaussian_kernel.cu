#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
 * example application.
 * Device code.
 */

#define COVARIANCE_DYNAMIC_RANGE 1E3

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "gaussian.h"

#define sdata(index)      CUT_BANK_CHECKER(sdata, index)

/*
 * Compute the multivariate mean of the FCS data
 */ 
__device__ void mvtmeans(float* fcs_data, int num_dimensions, int num_events, float* means) {
    // access thread id
    int tid = threadIdx.x;

    if(tid < num_dimensions) {
        means[tid] = 0.0;

        // Sum up all the values for the dimension
        for(int i=0; i < num_events; i++) {
            means[tid] += fcs_data[i*num_dimensions+tid];
        }

        // Divide by the # of elements to get the average
        means[tid] /= (float) num_events;
    }
}

__device__ void averageVariance(float* fcs_data, float* means, int num_dimensions, int num_events, float* avgvar) {
    // access thread id
    int tid = threadIdx.x;
    
    __shared__ float variances[NUM_DIMENSIONS];
    __shared__ float total_variance;
    
    // Compute average variance for each dimension
    if(tid < num_dimensions) {
        variances[tid] = 0.0;
        // Sum up all the variance
        for(int j=0; j < num_events; j++) {
            // variance = (data - mean)^2
            variances[tid] += (fcs_data[j*num_dimensions + tid])*(fcs_data[j*num_dimensions + tid]);
        }
        variances[tid] /= (float) num_events;
        variances[tid] -= means[tid]*means[tid];
    }
    
    __syncthreads();
    
    if(tid == 0) {
        total_variance = 0.0;
        for(int i=0; i<num_dimensions;i++) {
            ////printf("%f ",variances[tid]);
            total_variance += variances[i];
        }
        ////printf("\nTotal variance: %f\n",total_variance);
        *avgvar = total_variance / (float) num_dimensions;
        ////printf("Average Variance: %f\n",*avgvar);
    }
}

// Inverts an NxN matrix 'data' stored as a 1D array in-place
// 'actualsize' is N
// Computes the log of the determinant of the origianl matrix in the process
__device__ void invert(float* data, int actualsize, float* log_determinant)  {
    int maxsize = actualsize;
    int n = actualsize;
    
    if(threadIdx.x == 0) {
        *log_determinant = 0.0;

      // sanity check        
      if (actualsize == 1) {
        *log_determinant = logf(data[0]);
        data[0] = 1.0 / data[0];
      } else {

          for (int i=1; i < actualsize; i++) data[i] /= data[0]; // normalize row 0
          for (int i=1; i < actualsize; i++)  { 
            for (int j=i; j < actualsize; j++)  { // do a column of L
              float sum = 0.0;
              for (int k = 0; k < i; k++)  
                  sum += data[j*maxsize+k] * data[k*maxsize+i];
              data[j*maxsize+i] -= sum;
              }
            if (i == actualsize-1) continue;
            for (int j=i+1; j < actualsize; j++)  {  // do a row of U
              float sum = 0.0;
              for (int k = 0; k < i; k++)
                  sum += data[i*maxsize+k]*data[k*maxsize+j];
              data[i*maxsize+j] = 
                 (data[i*maxsize+j]-sum) / data[i*maxsize+i];
              }
            }
            
            for(int i=0; i<actualsize; i++) {
                *log_determinant += logf(fabs(data[i*n+i]));
            }
            
          for ( int i = 0; i < actualsize; i++ )  // invert L
            for ( int j = i; j < actualsize; j++ )  {
              float x = 1.0;
              if ( i != j ) {
                x = 0.0;
                for ( int k = i; k < j; k++ ) 
                    x -= data[j*maxsize+k]*data[k*maxsize+i];
                }
              data[j*maxsize+i] = x / data[j*maxsize+j];
              }
          for ( int i = 0; i < actualsize; i++ )   // invert U
            for ( int j = i; j < actualsize; j++ )  {
              if ( i == j ) continue;
              float sum = 0.0;
              for ( int k = i; k < j; k++ )
                  sum += data[k*maxsize+j]*( (i==k) ? 1.0 : data[i*maxsize+k] );
              data[i*maxsize+j] = -sum;
              }
          for ( int i = 0; i < actualsize; i++ )   // final inversion
            for ( int j = 0; j < actualsize; j++ )  {
              float sum = 0.0;
              for ( int k = ((i>j)?i:j); k < actualsize; k++ )  
                  sum += ((j==k)?1.0:data[j*maxsize+k])*data[k*maxsize+i];
              data[j*maxsize+i] = sum;
              }
        }
    }
 }


__device__ void compute_pi(clusters_t* clusters, int num_clusters) {
    __shared__ float sum;
    
    if(threadIdx.x == 0) {
        sum = 0.0;
        for(int i=0; i<num_clusters; i++) {
            sum += clusters->N[i];
        }
    }
    
    __syncthreads();
    
    for(int c=threadIdx.x; c < num_clusters; c += blockDim.x) {
        if(clusters->N[c] < 0.5f) {
            clusters->pi[threadIdx.x] = 1e-10;
        } else {
            clusters->pi[threadIdx.x] = clusters->N[c] / sum;
        }
    }
 
    __syncthreads();
}


__device__ void compute_constants(clusters_t* clusters, int num_clusters, int num_dimensions) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int num_elements = num_dimensions*num_dimensions;
    
    __shared__ float determinant_arg; // only one thread computes the inverse so we need a shared argument
    
    float log_determinant;
    
    __shared__ float matrix[NUM_DIMENSIONS*NUM_DIMENSIONS];
    
    // Invert the matrix for every cluster
    int c = blockIdx.x;
    // Copy the R matrix into shared memory for doing the matrix inversion
    for(int i=tid; i<num_elements; i+= num_threads ) {
        matrix[i] = clusters->R[c*num_dimensions*num_dimensions+i];
    }
    
    __syncthreads(); 
    #if DIAG_ONLY
        if(tid == 0) { 
            determinant_arg = 1.0f;
            for(int i=0; i < num_dimensions; i++) {
                determinant_arg *= matrix[i*num_dimensions+i];
                matrix[i*num_dimensions+i] = 1.0f / matrix[i*num_dimensions+i];
            }
            determinant_arg = logf(determinant_arg);
        }
    #else 
        invert(matrix,num_dimensions,&determinant_arg);
    #endif
    __syncthreads(); 
    log_determinant = determinant_arg;
    
    // Copy the matrx from shared memory back into the cluster memory
    for(int i=tid; i<num_elements; i+= num_threads) {
        clusters->Rinv[c*num_dimensions*num_dimensions+i] = matrix[i];
    }
    
    __syncthreads();
    
    // Compute the constant
    // Equivilent to: log(1/((2*PI)^(M/2)*det(R)^(1/2)))
    // This constant is used in all E-step likelihood calculations
    if(tid == 0) {
        clusters->constant[c] = -num_dimensions*0.5f*logf(2.0f*PI) - 0.5f*log_determinant;
    }
}

/*
 * Computes the constant, pi, Rinv for each cluster
 * 
 * Needs to be launched with the number of blocks = number of clusters
 */
__global__ void
constants_kernel(clusters_t* clusters, int num_clusters, int num_dimensions) {
    compute_constants(clusters,num_clusters,num_dimensions);
    
    __syncthreads();
    
    if(blockIdx.x == 0) {
        compute_pi(clusters,num_clusters);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! @param fcs_data         FCS data: [num_events]
//! @param clusters         Clusters: [num_clusters]
//! @param num_dimensions   Number of dimensions in an FCS event
//! @param num_clusters     Number of clusters
//! @param num_events       Number of FCS events
////////////////////////////////////////////////////////////////////////////////
__global__ void
seed_clusters( float* fcs_data, clusters_t* clusters, int num_dimensions, int num_clusters, int num_events) 
{
    // access thread id
    int tid = threadIdx.x;
    // access number of threads in this block
    int num_threads = blockDim.x;

    // shared memory
    __shared__ float means[NUM_DIMENSIONS];
    
    // Compute the means
    mvtmeans(fcs_data, num_dimensions, num_events, means);

    __syncthreads();
    
    __shared__ float avgvar;
    
    // Compute the average variance
    averageVariance(fcs_data, means, num_dimensions, num_events, &avgvar);
        
    int num_elements;
    int row, col;
        
    // Number of elements in the covariance matrix
    num_elements = num_dimensions*num_dimensions; 

    __syncthreads();

    float seed;
    if(num_clusters > 1) {
        seed = (num_events-1.0f)/(num_clusters-1.0f);
    } else {
        seed = 0.0;
    }
    
    // Seed the pi, means, and covariances for every cluster
    for(int c=0; c < num_clusters; c++) {
        if(tid < num_dimensions) {
            clusters->means[c*num_dimensions+tid] = fcs_data[((int)(c*seed))*num_dimensions+tid];
        }
          
        for(int i=tid; i < num_elements; i+= num_threads) {
            // Add the average variance divided by a constant, this keeps the cov matrix from becoming singular
            row = (i) / num_dimensions;
            col = (i) % num_dimensions;

            if(row == col) {
                clusters->R[c*num_dimensions*num_dimensions+i] = 1.0f;
            } else {
                clusters->R[c*num_dimensions*num_dimensions+i] = 0.0f;
            }
        }
        if(tid == 0) {
            clusters->pi[c] = 1.0f/((float)num_clusters);
            clusters->N[c] = ((float) num_events) / ((float)num_clusters);
            clusters->avgvar[c] = avgvar / COVARIANCE_DYNAMIC_RANGE;
        }
    }
}



__device__ float parallelSum(float* data, const unsigned int ndata) {
  const unsigned int tid = threadIdx.x;
  float t;

  __syncthreads();

  // Butterfly sum.  ndata MUST be a power of 2.
  for(unsigned int bit = ndata >> 1; bit > 0; bit >>= 1) {
    t = data[tid] + data[tid^bit];  __syncthreads();
    data[tid] = t;                  __syncthreads();
  }
  return data[tid];
}

///////////////////////////////////////////////////////////////////////////
// Parallel reduction, for when all you want is the sum of a certain
// quantity computed for every 1 to N.  CODE should be something in terms
// of n.  The resulting sum will be placed in RESULT.
// tmp_buff, base_off, RESULT, and n must be previously defined, however 
// they will be overwritten during the execution of the macro.
#define REDUCE(N, CODE, RESULT)                                \
base_off = 0;                                                  \
RESULT = 0.0f;                                                 \
while (base_off + BLOCK_SIZE < N) {                            \
  n = base_off + tid;                                          \
  tmp_buff[tid] = CODE;                                        \
  RESULT += parallelSum(tmp_buff, BLOCK_SIZE);                 \
  base_off += BLOCK_SIZE;                                      \
}                                                              \
n = base_off + tid;                                            \
if (n < N) {tmp_buff[tid] = CODE;}                             \
else {tmp_buff[tid] = 0.0f;}                                   \
RESULT += parallelSum(tmp_buff, BLOCK_SIZE);
///////////////////////////////////////////////////////////////////////////

__device__ void compute_indices(int num_events, int* start, int* stop) {
    // Break up the events evenly between the blocks
    int num_pixels_per_block = num_events / gridDim.x;
    // Make sure the events being accessed by the block are aligned to a multiple of 16
    num_pixels_per_block = num_pixels_per_block - (num_pixels_per_block % 16);
    
    *start = blockIdx.x * num_pixels_per_block + threadIdx.x;
    
    // Last block will handle the leftover events
    if(blockIdx.x == gridDim.x-1) {
        *stop = num_events;
    } else {
        *stop = (blockIdx.x+1) * num_pixels_per_block;
    }
}

__global__ void
estep1(float* data, clusters_t* clusters, int num_dimensions, int num_events) {
    
    // Cached cluster parameters
    __shared__ float means[NUM_DIMENSIONS];
    __shared__ float Rinv[NUM_DIMENSIONS*NUM_DIMENSIONS];
    float cluster_pi;
    float constant;
    const unsigned int tid = threadIdx.x;
 
    int start_index;
    int end_index;

    int c = blockIdx.y;

    compute_indices(num_events,&start_index,&end_index);
    
    float like;

    // This loop computes the expectation of every event into every cluster
    //
    // P(k|n) = L(x_n|mu_k,R_k)*P(k) / P(x_n)
    //
    // Compute log-likelihood for every cluster for each event
    // L = constant*exp(-0.5*(x-mu)*Rinv*(x-mu))
    // log_L = log_constant - 0.5*(x-u)*Rinv*(x-mu)
    // the constant stored in clusters[c].constant is already the log of the constant
    
    // copy the means for this cluster into shared memory
    if(tid < num_dimensions) {
        means[tid] = clusters->means[c*num_dimensions+tid];
    }

    // copy the covariance inverse into shared memory
    for(int i=tid; i < num_dimensions*num_dimensions; i+= NUM_THREADS_ESTEP) {
        Rinv[i] = clusters->Rinv[c*num_dimensions*num_dimensions+i]; 
    }
    
    cluster_pi = clusters->pi[c];
    constant = clusters->constant[c];

    // Sync to wait for all params to be loaded to shared memory
    __syncthreads();
    
    for(int event=start_index; event<end_index; event += NUM_THREADS_ESTEP) {
       like = 0.0f;
        // this does the loglikelihood calculation
        #if DIAG_ONLY
            for(int j=0; j<num_dimensions; j++) {
                like += (data[j*num_events+event]-means[j]) * (data[j*num_events+event]-means[j]) * Rinv[j*num_dimensions+j];
            }
        #else
            for(int i=0; i<num_dimensions; i++) {
                for(int j=0; j<num_dimensions; j++) {
                    like += (data[i*num_events+event]-means[i]) * (data[j*num_events+event]-means[j]) * Rinv[i*num_dimensions+j];
                }
            }
        #endif
        // numerator of the E-step probability computation
        clusters->memberships[c*num_events+event] = -0.5f * like + constant + logf(cluster_pi);
    }
}

__global__ void
estep2(float* fcs_data, clusters_t* clusters, int num_dimensions, int num_clusters, int num_events, float* likelihood) {
    float temp;
    float thread_likelihood = 0.0f;
    __shared__ float total_likelihoods[NUM_THREADS_ESTEP];
    float max_likelihood;
    float denominator_sum;
    
    // Break up the events evenly between the blocks
    int num_pixels_per_block = num_events / gridDim.x;
    // Make sure the events being accessed by the block are aligned to a multiple of 16
    num_pixels_per_block = num_pixels_per_block - (num_pixels_per_block % 16);
    int tid = threadIdx.x;
    
    int start_index;
    int end_index;
    start_index = blockIdx.x * num_pixels_per_block + tid;
    
    // Last block will handle the leftover events
    if(blockIdx.x == gridDim.x-1) {
        end_index = num_events;
    } else {
        end_index = (blockIdx.x+1) * num_pixels_per_block;
    }
    
    total_likelihoods[tid] = 0.0;

    // P(x_n) = sum of likelihoods weighted by P(k) (their probability, cluster[c].pi)
    //  log(a+b) != log(a) + log(b) so we need to do the log of the sum of the exponentials

    //  For the sake of numerical stability, we first find the max and scale the values
    //  That way, the maximum value ever going into the exp function is 0 and we avoid overflow

    //  log-sum-exp formula:
    //  log(sum(exp(x_i)) = max(z) + log(sum(exp(z_i-max(z))))
    for(int pixel=start_index; pixel<end_index; pixel += NUM_THREADS_ESTEP) {
        // find the maximum likelihood for this event
        max_likelihood = clusters->memberships[pixel];
        for(int c=1; c<num_clusters; c++) {
            max_likelihood = fmaxf(max_likelihood,clusters->memberships[c*num_events+pixel]);
        }

        // Compute P(x_n), the denominator of the probability (sum of weighted likelihoods)
        denominator_sum = 0.0;
        for(int c=0; c<num_clusters; c++) {
            temp = expf(clusters->memberships[c*num_events+pixel]-max_likelihood);
            denominator_sum += temp;
        }
        denominator_sum = max_likelihood + logf(denominator_sum);
        thread_likelihood += denominator_sum;
        
        // Divide by denominator, also effectively normalize probabilities
        // exp(log(p) - log(denom)) == p / denom
        for(int c=0; c<num_clusters; c++) {
            clusters->memberships[c*num_events+pixel] = expf(clusters->memberships[c*num_events+pixel] - denominator_sum);
            //printf("Probability that pixel #%d is in cluster #%d: %f\n",pixel,c,clusters->memberships[c*num_events+pixel]);
        }
    }
    
    total_likelihoods[tid] = thread_likelihood;
    __syncthreads();

    temp = parallelSum(total_likelihoods,NUM_THREADS_ESTEP);
    if(tid == 0) {
        likelihood[blockIdx.x] = temp;
    }
}


/*
 * Means kernel
 * MultiGPU version, sums up all of the elements, but does not divide by N. 
 * This task is left for the host after combing results from multiple GPUs
 *
 * Should be launched with [M x D] grid
 */
__global__ void
mstep_means(float* fcs_data, clusters_t* clusters, int num_dimensions, int num_clusters, int num_events) {
    // One block per cluster, per dimension:  (M x D) grid of blocks
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int c = blockIdx.x; // cluster number
    int d = blockIdx.y; // dimension number

    __shared__ float temp_sum[NUM_THREADS_MSTEP];
    float sum = 0.0f;
    
    for(int event=tid; event < num_events; event+= num_threads) {
        sum += fcs_data[d*num_events+event]*clusters->memberships[c*num_events+event];
    }
    temp_sum[tid] = sum;
    
    __syncthreads();

    // Reduce partial sums
    sum = parallelSum(temp_sum,NUM_THREADS_MSTEP);
    if(tid == 0) {
        clusters->means[c*num_dimensions+d] = sum;
    }
}

/*
 * Computes the size of each cluster, N
 * Should be launched with M blocks (where M = number of clusters)
 */
__global__ void
mstep_N(clusters_t* clusters, int num_dimensions, int num_clusters, int num_events) {
    
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int c = blockIdx.x;
 
    
    // Need to store the sum computed by each thread so in the end
    // a single thread can reduce to get the final sum
    __shared__ float temp_sums[NUM_THREADS_MSTEP];

    // Compute new N
    float sum = 0.0f;
    // Break all the events accross the threads, add up probabilities
    for(int event=tid; event < num_events; event += num_threads) {
        sum += clusters->memberships[c*num_events+event];
    }
    temp_sums[tid] = sum;
 
    __syncthreads();

    sum = parallelSum(temp_sums,NUM_THREADS_MSTEP);
    if(tid == 0) {
        clusters->N[c] = sum;
    }
}
   
/*
 * Computes the row and col of a square matrix based on the index into
 * a lower triangular (with diagonal) matrix
 * 
 * Used to determine what row/col should be computed for covariance
 * based on a block index.
 */
__device__ void compute_row_col(int n, int* row, int* col) {
    int i = 0;
    for(int r=0; r < n; r++) {
        for(int c=0; c <= r; c++) {
            if(i == blockIdx.y) {  
                *row = r;
                *col = c;
                return;
            }
            i++;
        }
    }
}
 
/*
 * Computes the covariance matrices of the data (R matrix)
 * Must be launched with a M x D*D grid of blocks: 
 *  i.e. dim3 gridDim(num_clusters,num_dimensions*num_dimensions)
 */
__global__ void
mstep_covariance1(float* fcs_data, clusters_t* clusters, int num_dimensions, int num_clusters, int num_events) {
    int tid = threadIdx.x; // easier variable name for our thread ID

    // Determine what row,col this matrix is handling, also handles the symmetric element
    int row,col,c;
    compute_row_col(num_dimensions, &row, &col);
    //row = blockIdx.y / num_dimensions;
    //col = blockIdx.y % num_dimensions;

    __syncthreads();
    
    c = blockIdx.x; // Determines what cluster this block is handling    

    int matrix_index = row * num_dimensions + col;

    #if DIAG_ONLY
    if(row != col) {
        clusters->R[c*num_dimensions*num_dimensions+matrix_index] = 0.0;
        matrix_index = col*num_dimensions+row;
        clusters->R[c*num_dimensions*num_dimensions+matrix_index] = 0.0;
        return;
    }
    #endif 

    // Store the means in shared memory to speed up the covariance computations
    __shared__ float means[NUM_DIMENSIONS];
    // copy the means for this cluster into shared memory
    if(tid < num_dimensions) {
        means[tid] = clusters->means[c*num_dimensions+tid];
    }

    // Sync to wait for all params to be loaded to shared memory
    __syncthreads();

    __shared__ float temp_sums[NUM_THREADS_MSTEP];
    
    float cov_sum = 0.0;

    for(int event=tid; event < num_events; event+=NUM_THREADS_MSTEP) {
        cov_sum += (fcs_data[row*num_events+event]-means[row])*(fcs_data[col*num_events+event]-means[col])*clusters->memberships[c*num_events+event]; 
    }
    temp_sums[tid] = cov_sum;

    __syncthreads();

    //cov_sum = parallelSum(temp_sums,NUM_THREADS);
    
    if(tid == 0) {
        cov_sum = 0.0;
        for(int i=0; i < NUM_THREADS_MSTEP; i++) {
            cov_sum += temp_sums[i];
        }
        if(clusters->N[c] >= 1.0) { // Does it need to be >=1, or just something non-zero?
            clusters->R[c*num_dimensions*num_dimensions+matrix_index] = cov_sum;
            // Set the symmetric value
            matrix_index = col*num_dimensions+row;
            clusters->R[c*num_dimensions*num_dimensions+matrix_index] = cov_sum;
        } else {
            clusters->R[c*num_dimensions*num_dimensions+matrix_index] = 0.0; // what should the variance be for an empty cluster...?
            // Set the symmetric value
            matrix_index = col*num_dimensions+row;
            clusters->R[c*num_dimensions*num_dimensions+matrix_index] = 0.0; // what should the variance be for an empty cluster...?
        }

        // Regularize matrix - adds some variance to the diagonal elements
        // Helps keep covariance matrix non-singular (so it can be inverted)
        // The amount added is scaled down based on COVARIANCE_DYNAMIC_RANGE constant defined at top of this file
        if(row == col) {
            clusters->R[c*num_dimensions*num_dimensions+matrix_index] += clusters->avgvar[c];
        }
    }
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
